#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>
#include <bits/stdc++.h>

#include <climits>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>



#include <chrono>  //this is from https://stackoverflow.com/questions/22387586/measuring-execution-time-of-a-function-in-c

using namespace std;
using namespace std::chrono;


__global__ void line89 (int *d_dstp, int *d_srcp, int n){
  int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index >=0 && index < n)
    d_dstp [ d_srcp[index] ] =index;

}



int main(int argc, char *argv[])
{
    ifstream fin;
    ofstream fout;
    string infile, outfile;
    char* s=argv[1];
    char* t=argv[2];
    cout<<"inside readGraph"<<endl;
  // infile ="../../../input/"      + name + ".mmio" ; //  ../../../input/amazon0302_adj.mmio
  // outfile="../../output/serial/" + name + ".txt"  ; //  dataset+"-out.txt";
  infile =s;
  outfile = t;
  fin.open(infile.c_str());    // opening the input file
  fout.open(outfile.c_str());  // opening the output file

  string temp;
  getline(fin,temp); // readint the description line 1
  getline(fin,temp); // reading the description line 2

  //int temp_e;          // temperory edge because edge weight is useless
  int u1,v1,n,m;             // the v1,v2 of edges

  fin >> n >> n >> m ;       // reading the MxN graph and edges
  cout<< n<<" "<< m<<endl;

  //int hist[n], srcp[n], dstp[n];

  thrust::device_vector<int> hist(n);
  thrust::device_vector<int> srcp(n);
  thrust::device_vector<int> dstp(n);

  int i,temp_int;

  high_resolution_clock::time_point t1 = high_resolution_clock::now();
  thrust::fill(hist.begin(), hist.begin() + n, 0);

  for(i=0;i<m;i++)
  {
      fin>>u1>>v1>>temp_int;
      hist[u1]++;
      hist[v1]++;

  }
  for( i = 0; i < hist.size(); i++)
        cout << "hist[" << i << "] = " << hist[i] << endl;

    thrust::sequence(srcp.begin(), srcp.end(),0);

    for(i = 0; i < srcp.size(); i++)
    cout << "srcp[" << i << "] = " << srcp[i] << endl;

    thrust::sort_by_key(hist.begin(), hist.begin() + n, srcp.begin(),thrust::greater<int>());

    for(i = 0; i < srcp.size(); i++)
    cout << "srcp[" << i << "] = " << srcp[i] << endl;

    // for(i=0;i<n;i++)
    // {
    //     dstp[srcp[i]]=i;
    // }

    int *d_dstp = thrust::raw_pointer_cast(&dstp[0]);
    int *d_srcp = thrust::raw_pointer_cast(&srcp[0]);
    line89<<< n/1024 + 1 , 1024>>>(d_dstp, d_srcp, n);


    thrust::device_vector<int> u(m);
    thrust::device_vector<int> v(m);

    fin.close();
    fin.open(infile.c_str());    // opening the input file
    // fout.open(outfile.c_str());  // opening the output file

    getline(fin,temp); // readint the description line 1
    getline(fin,temp); // reading the description line 2

    //int temp_e;          // temperory edge because edge weight is useless
    //int u1,v1,n,m;             // the v1,v2 of edges

    fin >> n >> n >> m ;
    //set< pair<int,int> > st;
    int u_,v_;

    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    thrust::host_vector< thrust::pair<int,int> > h_E(m);
    for(i=0;i<m;i++)
    {
        fin>>u1>>v1>>temp_int;
        u_ = dstp[u1];
        v_ = dstp[v1];
        h_E[i] = ( u_>v_ ? thrust::make_pair(u_,v_) : thrust::make_pair(v_,u_));//  make_pair(max(u_,v_),max(u_,v_)));
    }

    // std::set<int>::iterator it;
    // for(it = st.begin(); it != st.end(); it++){

    // }
    thrust::host_vector< thrust::pair<int,int> > d_E (h_E.begin(),h_E.end());

    thrust::sort(d_E.begin(), d_E.end());
    bool flag1=false;
    if(d_E[0].first == d_E[1].first && d_E[0].second == d_E[1].second)
      flag1 = true;

    auto new_E = thrust::unique(d_E.begin(),d_E.end());
    high_resolution_clock::time_point t3 = high_resolution_clock::now();
    //https://stackoverflow.com/questions/49856303/thrustunique-on-float3-tuple

    auto a = duration_cast<microseconds>( t2 - t1 ).count();
    auto b = duration_cast<microseconds>( t3 - t2 ).count();
    auto c = duration_cast<microseconds>( t3 - t1 ).count();
    fout<<n<<" "<<m<<endl;
    cout<<"a "<<a<<" b "<<b<<" c "<<c<<endl;
    if(flag == true)
      for(int i=0 ; i<(d_E.size()/2) ; i++)
        fout<<d_E[i].first << " "<<d_E[i].second<<endl;
    else
      for(int i=0 ; i<(d_E.size()) ; i++)
        fout<<d_E[i].first << " "<<d_E[i].second<<endl;
    fout.close();



return 0;


}
