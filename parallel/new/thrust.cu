#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>
#include <bits/stdc++.h>

#include <climits>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>



using namespace std;

__global__ void line89 (int *d_dstp, int *d_srcp, int n){
  int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index >=0 && index < n)
    d_dstp [ d_srcp[index] ] =index;

}

int main()
{
    ifstream fin;
    ofstream fout;
    string infile, outfile;

    cout<<"inside readGraph"<<endl;
  // infile ="../../../input/"      + name + ".mmio" ; //  ../../../input/amazon0302_adj.mmio
  // outfile="../../output/serial/" + name + ".txt"  ; //  dataset+"-out.txt";
  infile ="ip.txt";

  fin.open(infile.c_str());    // opening the input file
  fout.open(outfile.c_str());  // opening the output file

  string temp;
  getline(fin,temp); // readint the description line 1
  getline(fin,temp); // reading the description line 2

  //int temp_e;          // temperory edge because edge weight is useless
  int u1,v1,n,m;             // the v1,v2 of edges

  fin >> n >> n >> m ;       // reading the MxN graph and edges
  cout<< n<<" "<< m<<endl;

  //int hist[n], srcp[n], dstp[n];

  thrust::device_vector<int> hist(n);
  thrust::device_vector<int> srcp(n);
  thrust::device_vector<int> dstp(n);

  int i;

  thrust::fill(hist.begin(), hist.begin() + n, 0);

  for(i=0;i<m;i++)
  {
      fin>>u1>>v1;
      hist[u1]++;
      hist[v1]++;

  }
  // for( i = 0; i < hist.size(); i++)
  //       cout << "hist[" << i << "] = " << hist[i] << endl;

    thrust::sequence(srcp.begin(), srcp.end(),0);

    // for(i = 0; i < srcp.size(); i++)
    // cout << "srcp[" << i << "] = " << srcp[i] << endl;

    thrust::sort_by_key(hist.begin(), hist.begin() + n, srcp.begin(),thrust::greater<int>());

    // for(i = 0; i < srcp.size(); i++)
    // cout << "srcp[" << i << "] = " << srcp[i] << endl;

    // for(i=0;i<n;i++)
    // {
    //     dstp[srcp[i]]=i;
    // }

    int *d_dstp = thrust::raw_pointer_cast(&dstp[0]);
    int *d_srcp = thrust::raw_pointer_cast(&srcp[0]);
    line89<<< n/1024 + 1 , 1024>>>(d_dstp, d_srcp, n);
    for(i = 0; i < dstp.size(); i++)
      cout << "dstp[" << i << "] = " << dstp[i] << endl;
    thrust::device_vector<int> u(m);
    thrust::device_vector<int> v(m);
    

return 0;


}
