#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>
#include <bits/stdc++.h>
#include <algorithm>

#include <climits>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>



using namespace std;



int main()
{
    ifstream fin;
    ofstream fout;
    string infile, outfile;

    cout<<"inside readGraph"<<endl;
  // infile ="../../../input/"      + name + ".mmio" ; //  ../../../input/amazon0302_adj.mmio
  // outfile="../../output/serial/" + name + ".txt"  ; //  dataset+"-out.txt";
  infile ="ip.txt";

  fin.open(infile.c_str());    // opening the input file
  fout.open(outfile.c_str());  // opening the output file

  string temp;
  getline(fin,temp); // readint the description line 1
  getline(fin,temp); // reading the description line 2

  //int temp_e;          // temperory edge because edge weight is useless
  int u,v,n,m;             // the v1,v2 of edges

  fin >> n >> n >> m ;       // reading the MxN graph and edges
  cout<< n<<" "<< m<<endl;

  //int hist[n], srcp[n], dstp[n];

  thrust::device_vector<int> hist(n);
  thrust::device_vector<int> srcp(n);
  thrust::device_vector<int> dstp(n);

  int i;

  thrust::fill(hist.begin(), hist.begin() + n, 0);

  for(i=0;i<m;i++)
  {
      fin>>u>>v;
      hist[u]++;
      hist[v]++;

  }
  for( i = 0; i < hist.size(); i++)
        cout << "hist[" << i << "] = " << hist[i] << endl;

    thrust::sequence(srcp.begin(), srcp.end(),0);

    for(i = 0; i < srcp.size(); i++)
    cout << "srcp[" << i << "] = " << srcp[i] << endl;

    thrust::sort_by_key(hist.begin(), hist.begin() + n, srcp.begin(),thrust::greater<int>());

    for(i = 0; i < srcp.size(); i++)
    cout << "srcp[" << i << "] = " << srcp[i] << endl;

    for(i=0;i<n;i++)
    {
        dstp[srcp[i]]=i;
    }

    fin.close();
    fin.open(infile.c_str());
    getline(fin,temp);        // readint the description line 1
    getline(fin,temp);        // reading the description line 2
    fin >> n >> n >> m ;      // reading the MxN graph and edges
    cout<< n<<" "<< m<<endl;
    int u_,v_;
    for(i=0;i<m;i++){
        fin>>u>>v;
        u_ = dstp[u];
        v_ = dstp[v];
        
    }



return 0;


}
