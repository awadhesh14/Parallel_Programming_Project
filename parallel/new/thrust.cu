#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>

#include <climits>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


using namespace std;

ifstream fin;
ofstream fout;
string infile, outfile;

int main()
{
    cout<<"inside readGraph"<<endl;
  // infile ="../../../input/"      + name + ".mmio" ; //  ../../../input/amazon0302_adj.mmio
  // outfile="../../output/serial/" + name + ".txt"  ; //  dataset+"-out.txt";
  infile ="../../test_dir.txt";

  fin.open(infile.c_str());    // opening the input file
  fout.open(outfile.c_str());  // opening the output file

  string temp;
  getline(fin,temp); // readint the description line 1
  getline(fin,temp); // reading the description line 2

  int temp_e;          // temperory edge because edge weight is useless
  int u,v;             // the v1,v2 of edges

  fin >> g->n >> g->n >> g->m ;       // reading the MxN graph and edges
  cout<< g->n<<" "<< g->m<<endl;

  //int hist[n], srcp[n], dstp[n];

  thrust::host_vector<int> hist(g->n);

  int i;
  for(i=0;i<m;i++)
  {
      fin>>u>>v;
      hist[u]++;
      hist[v]++;
  }
  for(int i = 0; i < H.size(); i++)
        cout << "hist[" << i << "] = " << hist[i] << endl;

return 0;


}

