#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>

#include <climits>
#include<hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <math.h>


using namespace std;

typedef unsigned int eid_t;
typedef long var;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define GS 1024
#define BS 1024

typedef struct {
    var n; //N vertices
    var m; //M edges
    var num_of_rows; //n no of nonempty rows

    eid_t *rows;  //rows [n]
    eid_t *adj;   //cols [NNZ] = M (initially)
    eid_t *num_edges; //roff [N+1]
    eid_t *rlen;  //rlen [N]

} graph_t;

void free_graph(graph_t *g) {
    if( g->adj != NULL )
	free( g->adj );

    if( g->num_edges != NULL )
	free( g->num_edges );

    }


static double timer() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) (tp.tv_sec) + tp.tv_usec * 1e-6);
}

/*********************** READ INPUT FILE  ************************************************************/

int load_graph_from_file(char *filename, graph_t *g) {



    FILE *infp = fopen(filename, "r");
    if (infp == NULL) {
        fprintf(stderr, "Error: could not open inputh file: %s.\n Exiting ...\n", filename);
        exit(1);
    }

    fprintf(stdout, "Reading input file: %s\n", filename);

    //double t0 = timer();

    //Read N and M
    fscanf(infp, "%ld %ld\n", &(g->n), &(g->m));
    printf("N: %ld, M: %ld \n", g->n, g->m);

    bool flag[g->n];
    var m = 0;

    //Allocate space
    g->num_edges = (eid_t *) malloc((g->n + 1) * sizeof(eid_t));
    assert(g->num_edges != NULL);

    var i ;
    for ( i=0; i<g->n + 1; i++) {
        g->num_edges[i] = 0;
    }


    for ( i=0; i<g->n; i++) {
        flag[i] = false;
    }

    eid_t u, v;
   printf(" Checking1\n ");
    while( fscanf(infp, "%u %u\n", &u, &v) != EOF ) {

        if (u>v)
          { g->num_edges[u]++; flag[u] = true; m++; }
        else if(u<v)
          { g->num_edges[v]++; flag[v] = true; m++;}
    }
   printf(" Checking2\n ");

  g->rows = (eid_t *) malloc((g->n) * sizeof(eid_t));
  g->num_of_rows = 0;

  var k =0;
  for (i = 0; i<g->n; i++)
   { if (flag[i] == true)
           { g->num_of_rows++;
             g->rows[k] = i;
             k++;
           }
   }


  g->m = m;

  /*cout<<"flag"<<endl;

     for(long i = 0; i <  g->n; i++)
     {        cout<<flag[i] <<endl;
      }
  cout<<endl;

 cout<<"g->rows"<<endl;

      for(long i = 0; i < g->num_of_rows; i++)
     {        cout<<g->rows[i] <<endl;
      }
  cout<<endl;




 /* cout<<"num edges"<<endl;

      for(long i = 0; i <  m; i++)
     {        cout<<g->num_edges[i] <<endl;
      }
  cout<<endl;
  */

    fclose( infp );

   /* if( m != g->m) {
        printf("Reading error: file does not contain %ld edges.\n", g->m);
        free( g->num_edges );
        exit(1);
    }
   */

    m = 0;

    eid_t *temp_num_edges = (eid_t *) malloc((g->n + 1) * sizeof(eid_t));
    assert(temp_num_edges != NULL);

    temp_num_edges[0] = 0;

    for(i = 0; i < g->n; i++) {
        m += g->num_edges[i];
        temp_num_edges[i+1] = m;
    }




    //Allocate space for adj
    g->adj = (eid_t *) malloc(m * sizeof(eid_t));
    assert(g->adj != NULL);


   for(i= 0; i < g->n+1; i++)
            g->num_edges[i] = temp_num_edges[i];

  /*
  cout<<"g->num edges"<<endl;

      for(long i = 0; i <  g->n+1; i++)
     {        cout<<g->num_edges[i] <<endl;
      }
  cout<<endl;
*/

   g->rlen = (eid_t *) malloc((g->n) * sizeof(eid_t));

   k =0;

   for ( i = 0; i<g->n; i++)
   { if (flag[i] == true)
           { g->rlen[k] = g->num_edges[i+1] - g->num_edges[i];
           }
     else
           g->rlen[k] = 0;
     k++;
   }





    infp = fopen(filename, "r");
    if (infp == NULL) {
        fprintf(stderr, "Error: could not open input file: %s.\n Exiting ...\n", filename);
        exit(1);
    }

    //Read N and M
    fscanf(infp, "%ld %ld\n", &(g->n), &m);

   for(i = 0; i < m; i++)
            g->adj[i] = 0;


    //Read the edges
    while( fscanf(infp, "%u %u\n", &u, &v) != EOF ) {
        if ( u > v )

           { g->adj[ temp_num_edges[u]  ] = v;
             temp_num_edges[u]++;

           }

        else if (u<v)
           {
            g->adj[ temp_num_edges[v] ] = u;
            temp_num_edges[v]++;

           }


    }


  fclose( infp );
/*
  cout<<" g->adj"<<endl;
     for(long i = 0; i <  m; i++)
     {        cout<<g->adj[i] <<endl;
      }


   cout<<" g->rlen"<<endl;
     for(long i = 0; i <  g->n; i++)
     {        cout<<g->rlen[i] <<endl;
      }

*/



    //free( temp_num_edges );
    return 0;
}





//***************************************************** CUDA KERNEL *****************************************************************

__global__ void support_compute(eid_t* roff, eid_t* rows, eid_t *cols,  int* bitmap, eid_t* rlen, var M, var N, var m, int* supp, int k, bool* weak, bool* weak_vertices)

{
   //printf ("Inside Kernel \n");
   __shared__ int value[BS];

    int tid = threadIdx.x;

    eid_t io_s, io_e, io, jo_s, jo_e, jo, i;
    int c;

for(var s = blockIdx.x; s<M; s += gridDim.x)
 {

        i = rows[s];
	io_s = roff[i];
        io_e = io_s + rlen[i];

	for(io = io_s; io<io_e; io +=blockDim.x)
 	{
           value[tid] = -1;
           c= -1;
	   c = ((io+threadIdx.x)<io_e) ? (cols[io+threadIdx.x]): -1;

           if (c>-1)
	      { atomicOr((bitmap+(N*blockIdx.x)+c), 1);
                value[tid] = c;
              }

           __syncthreads();


            for(int t=0; t<blockDim.x; t++)
	     {

                 int j = value[t];
                 if(j == -1)
                    break;


                 var cnt = 0;
	         jo_s = roff[j];
		 jo_e = jo_s + rlen[j];


  		 for(jo = jo_s + threadIdx.x; jo < jo_e; jo += blockDim.x)
		 {
			       eid_t k = cols[jo];
			        if (*(bitmap+(blockIdx.x*N)+k) == 1)
			          { cnt++;
                                    atomicAdd(supp + jo, 1);
                                    eid_t a=0;
                                      for( a =0; a <= rlen[i]; a++)
                                       {  if (cols[io_s + a] == k)
                                              break;

                                       }


                                    atomicAdd(supp+io_s+a, 1);

                                  }

	         }


                 atomicAdd(supp+io+t, cnt);

	    }



        }
        atomicAnd((bitmap+(N*blockIdx.x))+c, 0);



 }

 //End of support computation

  __syncthreads();

/*
if (threadIdx.x == 0 && blockIdx.x == 0)
 { printf("Support\n");

 for( int i = 0; i<m; i++)
        printf("%d \n", supp[i]);
 }



*/




 __shared__ int flag;

 while(true)
  { //atomicAnd(point, 0);
    //flag[threadIdx.x] = 0;

    flag = 0;
    for (int s = blockIdx.x; (s*blockDim.x + threadIdx.x)<m; s+= gridDim.x)
     {
       int i = s* blockDim.x + threadIdx.x;
 //printf("it = %d, blockId =  %d, threadId =  %d, s = %d, i = %d,  weak[i]= %d, supp[i] = %d\n", it, blockIdx.x , threadIdx.x, s, i, weak[i], supp[i]);
       if (supp[i] < k-2 && weak[i] == 0)
        {
          weak[i] = 1;
          supp[i] = -1;
          flag = 1;

        }


     }
    __syncthreads();




   if(flag == 0)
      break;

    __syncthreads();



   if(k>3)
     { for (int s = blockIdx.x; (s*blockDim.x + threadIdx.x)<m; s+= gridDim.x)
        {
            int i = s* blockDim.x + threadIdx.x;

       		if (weak[i] == 1)
        	{ int j=0;

                  long start = 0, end = N, mid;
                  while(start<end)
                 { mid = (start+end)/2;
                   j = mid;
                   if(i+1 > roff[mid] ) start = mid+1;
                   else end = mid-1;


                 }


                  weak_vertices[j] = 1;

        	}

     	}
      __syncthreads();

     for (int s = blockIdx.x; (s*blockDim.x + threadIdx.x)<N; s+= gridDim.x)
        {
            int i = s* blockDim.x + threadIdx.x;

       		if (weak_vertices[i] == 1)
        	{ int j=0;
                  for ( j = 0; j<rlen[i]; j++)
                     { int u = roff[i+j];
                       weak_vertices[cols[u]] = 1;
                       //printf("blockId =  %d, threadId =  %d, u = %d \n", blockIdx.x , threadIdx.x, cols[u]);
                     }
                 }

     	}
      __syncthreads();



     for(int s = blockIdx.x; s<M; s += gridDim.x)
 	{
        	i = rows[s];
		io_s = roff[i];
        	io_e = io_s + rlen[i];

                if(weak_vertices[i] == 1)
		{
                for(io = io_s; io<io_e; io +=blockDim.x)
 		{
           		value[tid] = -1;
           		c= -1;
	   		c = ((io+threadIdx.x)<io_e) ? (cols[io+threadIdx.x]): -1;

           		if (c>-1)
	      			{ atomicOr((bitmap+(N*blockIdx.x)+c), 1);
               			  value[tid] = c;
              			}

           		__syncthreads();

            		for(int t=0; t<blockDim.x; t++)
	     			{
                 			int j = value[t];
                 			if(j == -1)
                    				break;

                                        if (weak_vertices[j] == 1)
                 			{
                                        int cnt = 0;
	         			jo_s = roff[j];
		 			jo_e = jo_s + rlen[j];


		 			for(jo = jo_s + threadIdx.x; jo < jo_e; jo += blockDim.x)
		 			{
					       eid_t k = cols[jo];
					       if (*(bitmap+(blockIdx.x*N)+k) == 1)
					          { cnt++;


                 		                   eid_t a=0;
                 		                   for( a =0; a<=rlen[i]; a++)
                 		                      { if (cols[io_s + a] == k)
                 		                              break;
                 		                      }




                                                    if(weak_vertices[k] == 1)
                 	                                 {

                                                               if( supp[jo] == -1 || supp[io_s+a] == -1 || supp[io+t] == -1)
                                                                    { if ( supp[jo] != -1) atomicSub(supp+jo, 1);
                                                                      if ( supp[io_s+a] != -1) atomicSub(supp+io_s+a, 1);
                                                                      if ( supp[io+t] != -1) atomicSub(supp+io+t, 1);

                                                                    }
                                                                 //printf("i =  %d, j =  %d, k = %d, f = %d\n", i, j, k,f);


                                                         }//if k end
                 		                  }
	         			}


                  		       //printf("i =  %d, j =  %d, cnt = %d, i,j = %d \n", i, j, cnt, io+t);

	    		           }//if j end
				}


        	}
        	atomicAnd((bitmap+(N*blockIdx.x))+c, 0);

            }//if i end
       }//for end

      __syncthreads();

     }  //(k > 3) end

 if (k==3) break;


 } // while end




}// cuda end




int main(int argc, char *argv[]) {

    graph_t g;
    int gs=GS;
    int k = 68;

  //   if( argc < 2 )
  //   {
	// fprintf(stderr, "%s <Graph file>\n", argv[0]);
	// exit(1);
  //   }


        load_graph_from_file(/*argv[1]*/"../../test_dir.txt", &g);
        cout<<"File read complete"<<endl;


     int *b;
     b= (int *) malloc(g.m*sizeof(int));

     for(int i=0;i<g.m;i++)
         *(b + i ) = 0;


     int *bm = (int *) malloc((g.n)*gs*sizeof(int));

     for(int i=0;i<gs;i++)
     	{
		for(int j=0;j<(g.n);j++)
         	{
		  *(bm + i*g.n +j)=0;

		}
	}

	cout<<"g.num_of_rows = "<<g.num_of_rows<<endl;
   int *supp;
   eid_t *roff;
   eid_t *r;
   eid_t *col;
   eid_t* rl;
   int *bitmap;
   bool *weak, *wh;
   bool* weak_vertices;

   wh= (bool *) malloc(g.m*sizeof(bool));
     for(int i=0;i<g.m;i++)
         *(wh + i ) = false;

    cout<<" Malloc startng "<<endl;


    gpuErrchk(hipMalloc(&roff, (g.n + 1)*sizeof(eid_t)));
    gpuErrchk(hipMemcpy(roff, g.num_edges, (g.n + 1)*sizeof(eid_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&r, (g.num_of_rows)*sizeof(eid_t)));
    gpuErrchk(hipMemcpy(r, g.rows, ( g.num_of_rows )*sizeof(eid_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&col, (g.m)*sizeof(eid_t)));
    gpuErrchk(hipMemcpy(col, g.adj, (g.m)*sizeof(eid_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&bitmap, (g.n)*gs*sizeof(int)));
    gpuErrchk(hipMemcpy(bitmap, bm ,(g.n)*gs*sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&supp, g.m*sizeof(int)));
    gpuErrchk(hipMemcpy(supp, b, g.m*sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&rl, g.n*sizeof(eid_t)));
    gpuErrchk(hipMemcpy(rl, g.rlen, g.n*sizeof(eid_t), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&weak, g.m*sizeof(bool)));
    gpuErrchk(hipMemcpy(weak, wh, g.m*sizeof(bool), hipMemcpyHostToDevice));


    gpuErrchk(hipMalloc(&weak_vertices, g.n*sizeof(bool)));

        cout<<"sending into cuda"<<endl;

    	double t0 = timer();
        support_compute<<<GS,BS>>>(roff, r, col, bitmap, rl, g.num_of_rows, g.n, g.m, supp, k, weak, weak_vertices);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        cout<<"Time: "<< timer() - t0<<" sec\n ";



        gpuErrchk(hipMemcpy(b, supp, g.m*sizeof(int), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(wh, weak, g.m*sizeof(bool), hipMemcpyDeviceToHost));

        cout<<"Return from Cuda"<<endl;



/*
  printf("Support\n");

 for(  var i = 0; i<g.m; i++)
        printf("%d \n", b[i]);




  printf("Weak\n");

 for( int i = 0; i<g.m; i++)
        printf("%d \n", wh[i]);
*/


 return 0;
}
