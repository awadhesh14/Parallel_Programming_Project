//scp -r /home/awd/work/coursework/DS295/project/pp_project/parallel/* dtanwar@turing-gpu.cds.iisc.ac.in:/home/dtanwar/Project/Parallel_Programming_Project/parallel

#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>

#include <fstream>


#include <climits>
#include<hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <math.h>


#define GS 1024
#define BS 1024

using namespace std;

typedef unsigned int uui;
typedef int var;


typedef struct{
  var V;  //no of vertices
  var E;  //no of edges
  var n; //no of non empty rows
  //var E;

  uui *colind;  //nonzeroes in each row (colind)
  uui *roff;  //startig offset of each row (rowoff)
  uui *rlen;  //length of each row
  uui *rows;  //indices of the non empty rows
} G;

// #include "read_graph.hpp"
ifstream fin;
ofstream fout;
string infile, outfile;
void readGraph(string filename, G *g){
  // cout<<"inside readGraph"<<endl;
  // infile ="../../../input/"      + name + ".mmio" ; //  ../../../input/amazon0302_adj.mmio
  // outfile="../../output/serial/" + name + ".txt"  ; //  dataset+"-out.txt";
  infile =filename;

  fin.open(infile.c_str());    // opening the input file
  fout.open(outfile.c_str());  // opening the output file

  string temp;
  getline(fin,temp); // readint the description line 1
  getline(fin,temp); // reading the description line 2

  var temp_edge;          // temperory edge because edge weight is useless
  var u,v;             // the v1,v2 of edges

  fin >> g->V >> g->V >> g->E ;       // reading the MxN graph and edges
  //cout<< g->V<<" "<< g->E<<endl;      // just checking if it worked



/**************************allocating & initializing all flag[V] to false**********************************/
  bool flag[g->V];                // tells whether particular row is empty or not
  for (var i=0 ; i < g->V ; i++) {
      flag[i] = false;            // false means empty
  }

/**************************allocating & initializing all roff[V+1] to zero**********************************/
  g->roff = (uui *) malloc((g->V + 1) * sizeof(uui));
  assert(g->roff != NULL);
  for (var i=0 ; i < g->V+1 ; i++) {
      g->roff[i] = 0;
      //cout<<g->roff[i]<<" ";
  };
  //cout<<endl;

/**************************increase row offset and set flag for non empty row********************************/
	for (var i=0; i<g->E; ++i) {           //thrust
		fin >> u >> v >>temp_edge;
    //cout<< u <<" "<<v <<endl;

    if(u > v)
      g->roff[u+1]++ , flag[u] = true;
    else if(u < v)
      g->roff[v+1]++ , flag[v] = true;

	}

/**********************populates indexs of nonzero rows rows[n] and initilizes n (no of non empty rows)******/
  g->rows = (uui *) malloc((g->V) * sizeof(uui));
  g->n = 0;


  var k =0;
  for (var i = 0; i<g->V; i++){
     if (flag[i] == true){
       g->n++;                            //thrust
       g->rows[k++] = i;                    //thrust
     }
   }

/**********************************************************************************************************/
//converting the roff from degree holder to actual usage.
  uui *temp_num_edges = (uui *) malloc((g->V + 1) * sizeof(uui));
  assert(temp_num_edges != NULL);

  temp_num_edges[0] = 0;
  //g->E= 0;
  k=0;
  for(var i = 0; i < g->V; i++) {
    //  g->E += g->roff[i];
      k += g->roff[i+1];
      temp_num_edges[i+1] =k;
  }

  for(var i= 0; i < g->V+1; i++)
    g->roff[i] = temp_num_edges[i];

/**********************************************************************************************************/
  g->rlen = (uui *) malloc((g->V) * sizeof(uui));
  k =0;

  for (var i = 0; i<g->V; i++){
    if (flag[i] == true)
      g->rlen[k] = g->roff[i+1] - g->roff[i];
    else
      g->rlen[k] = 0;
    k++;
  }

/**********************************************************************************************************/
  //Allocate space for colind
  g->colind = (uui *) malloc(g->E * sizeof(uui));
  assert(g->colind != NULL);

  fin.close();
  fin.open(infile.c_str());
  getline(fin,temp); // readint the description line 1
  getline(fin,temp); // reading the description line 2

  //Read V and E
  //fscanf(infp, "%ld %ld\n", &(g->n), &g->E);
  fin>>(g->V)>>(g->V)>>g->E;
  for(var i = 0; i < g->E; i++)
    g->colind[i] = 0;
  //Read the edges
  // while( fscanf(infp, "%u %u\n", &u, &v) != EOF ) {
  for(var i=0 ; i<g->E ; i++){


    fin>>u>>v>>temp_edge;
    if(u>v){
      g->colind[ temp_num_edges[u]  ] = v;
      temp_num_edges[u]++;
    }
    else if (u<v){
      g->colind[ temp_num_edges[v] ] = u;
      temp_num_edges[v]++;
    }


  }
  fin.close();

/**********************************************************************************************************/

}



int main(int argc, char *argv[]){

  G g;
  int gs=GS;
  int k = 100;
  // cout<<endl<<"checkpoint 1"<<endl;
  readGraph("../../test_dir.txt",&g);
  // cout<<"checkpoint 2"<<endl;

  // cout<<"rows"<<endl;
  // for (var i=0;i<(g.n) ;i++){
  //   cout<<g.rows[i]<<" ";
  // }
  // cout<<endl;
  // cout<<"colind"<<endl;
  // for (var i=0;i<(g.E) ;i++){
  //   cout<<g.colind[i]<<" ";
  // }
  // cout<<endl;
  // cout<<"roff"<<endl;
  // for (var i=0;i<(g.V+1) ;i++){
  //   cout<<g.roff[i]<<" ";
  // }
  // cout<<endl;
  // cout<<"rlen"<<endl;
  // for (var i=0;i<(g.V) ;i++){
  //   cout<<g.rlen[i]<<" ";
  // }
  // cout<<endl;




}
