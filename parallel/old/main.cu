#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>

#include <fstream>


#include <climits>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>


#define GS 1024
#define BS 1024

using namespace std;

typedef unsigned int eid_t;
typedef long var;


typedef struct{
  var N;  //no of vertices
  var M;  //no of edges
  var n; //no of non empty rows
  var nnz;

  eid_t *cols;  //nonzeroes in each row (colind)
  eid_t *roff;  //startig offset of each row (rowoff)
  eid_t *rlen;  //length of each row
  eid_t *rows;  //indices of the non empty rows
} G;

#include "read_graph.hpp"


int main(int argc, char *argv[]){

  G g;
  int gs=GS;
  int k = 68;

  readGraph("../../test_dir.txt",&g);
  cout<<"checkpoint 1"<<endl;

  cout<<"rows"<<endl;
  for (var i=0;i<(g.n) ;i++){
    cout<<g.rows[i]<<" ";
  }
  cout<<endl;
  cout<<"cols"<<endl;
  for (var i=0;i<(g.nnz) ;i++){
    cout<<g.cols[i]<<" ";
  }
  cout<<endl;
  cout<<"roff"<<endl;
  for (var i=0;i<(g.N+1) ;i++){
    cout<<g.roff[i]<<" ";
  }
  cout<<endl;
  cout<<"rlen"<<endl;
  for (var i=0;i<(g.N) ;i++){
    cout<<g.rlen[i]<<" ";
  }
  cout<<endl;


}
