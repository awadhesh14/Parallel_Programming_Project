#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <functional>
#include <iostream>

#include <fstream>


#include <climits>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>


#define GS 1024
#define BS 1024

using namespace std;

typedef unsigned int eid_t;
typedef long var;


typedef struct{
  var N;  //no of vertices
  var M;  //no of edges
  var n; //no of non empty rows

  eid_t *cols;  //nonzeroes in each row (colind)
  eid_t *roff;  //startig offset of each row (rowoff)
  eid_t *rlen;  //length of each row
  eid_t *rows;  //indices of the non empty rows
} G;

#include "read_graph.hpp"


int main(int argc, char *argv[]){

  G g;
  int gs=GS;
  int k = 68;

  readGraph("../../test_dir.txt",&g);
  cout<<"checkpoint 1"<<endl;


}
